#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include "lzss.h"

#include "lzss.cpp"
static const int WORK_SIZE = 256;
FILE *infile, *outfile;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

__host__ __device__ unsigned int bitreverse(unsigned int number) {
	number = ((0xf0f0f0f0 & number) >> 4) | ((0x0f0f0f0f & number) << 4);
	number = ((0xcccccccc & number) >> 2) | ((0x33333333 & number) << 2);
	number = ((0xaaaaaaaa & number) >> 1) | ((0x55555555 & number) << 1);
	return number;
}

/**
 * CUDA kernel function that reverses the order of bits in each element of the array.
 */
__global__ void bitreverse(void *data) {
	unsigned int *idata = (unsigned int*) data;
	idata[threadIdx.x] = bitreverse(idata[threadIdx.x]);
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main_cuda(void) {
	void *d = NULL;
	int i;
	unsigned int idata[WORK_SIZE], odata[WORK_SIZE];

	for (i = 0; i < WORK_SIZE; i++)
		idata[i] = (unsigned int) i;

	CUDA_CHECK_RETURN(hipMalloc((void** ) &d, sizeof(int) * WORK_SIZE));
	CUDA_CHECK_RETURN(
			hipMemcpy(d, idata, sizeof(int) * WORK_SIZE,
					hipMemcpyHostToDevice));

	bitreverse<<<1, WORK_SIZE, WORK_SIZE * sizeof(int)>>>(d);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());
	CUDA_CHECK_RETURN(
			hipMemcpy(odata, d, sizeof(int) * WORK_SIZE,
					hipMemcpyDeviceToHost));

	for (i = 0; i < WORK_SIZE; i++)
		printf("Input value: %u, device output: %u, host output: %u\n",
				idata[i], odata[i], bitreverse(idata[i]));

	CUDA_CHECK_RETURN(hipFree((void* ) d));
	CUDA_CHECK_RETURN(hipDeviceReset());

	return 0;
}

/*
 * @TODO:
 int main(int argc, char *argv[])
 {
 char  *s;

 if (argc != 4) {
 printf("'lzss e file1 file2' encodes file1 into file2.\n"
 "'lzss d file2 file1' decodes file2 into file1.\n");
 return EXIT_FAILURE;
 }
 if ((s = argv[1], s[1] || strpbrk(s, "DEde") == NULL)
 || (s = argv[2], (infile  = fopen(s, "rb")) == NULL)
 || (s = argv[3], (outfile = fopen(s, "wb")) == NULL)) {
 printf("??? %s\n", s);  return EXIT_FAILURE;
 }
 if (toupper(*argv[1]) == 'E') Encode();  else Decode();
 fclose(infile);  fclose(outfile);
 return EXIT_SUCCESS;
 }
 */

/**
 * MAIN!
 */
int main(int argc, char *argv[]) {
	int enc;
	char *s;
	clock_t time = clock();
	if (argc != 4) {
		printf("Usage: lzss e/d infile outfile\n\te = encode\td =decode\n");
		return 1;
	}

	s = argv[1];
	if (s[1] == 0 && (*s == 'd' || *s == 'D' || *s == 'e' || *s == 'E'))
		enc = (*s == 'e' || *s == 'E');
	else {
		printf("? %s\n", s);
		return 1;
	}
	if ((infile = fopen(argv[2], "rb")) == NULL) {
		printf("? %s\n", argv[2]);
		return 1;
	}
	if ((outfile = fopen(argv[3], "wb")) == NULL) {
		printf("? %s\n", argv[3]);
		return 1;
	}


	DefaultLZSS defaultLZSS;
	//LZSS *lzss = &defaultLZSS;

	if (enc)
		defaultLZSS.encode(infile, outfile);
	else
		defaultLZSS.decode(infile, outfile);
	fclose(infile);
	fclose(outfile);
	printf("time: %.2f \n", (double) (clock() - time) / CLOCKS_PER_SEC);
	return 0;
}

